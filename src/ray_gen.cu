#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <optix_device.h>
//#include <units.h>
//#include <hiprand_uniform.h>
#include "transducer.h"
#include "helpers.h"
#include "ray.h"

//#define MAX_STACK_SIZE 10
#define EMPTY_STACK -1

using namespace optix;

struct PRD{
    float3 origin; //hit_point antes. Origen del rayo. Podria no usar mas rtCurrentRay
    float3 direction; //Direccion del rayo
    float importance; //intensity
    unsigned int depth;
    float distance_traveled;//units::length::millimeter_t distance_traveled;
    float time_traveled;
    float media_impedance;
    float media_attenuation;
    float media_mu0;
    float media_mu1;
    float media_sigma;
    float btti; //Back-To-Transducer Intensity
    int bounce_or_transmission; //0=bounce, 1=transmission
    int sample;
    int mesh_index;
    float is_vascular;
};



using namespace units::literals;

rtDeclareVariable(uint,     max_depth, , "Max amount of bounces");
rtDeclareVariable(uint,     max_rows, , "Max number of rows in output_buffer");
rtDeclareVariable(float,    scene_epsilon, , "Minimal importance admited.");
rtDeclareVariable(float,    axial_resolution, , );
rtDeclareVariable(float,    speed_of_sound, , );
rtDeclareVariable(float3,   bad_color, , "Exception color.");
rtDeclareVariable(rtObject, top_object, ,"Scene.");
rtDeclareVariable(float,    transducer_frequency, , "Transducer frequency.");
rtDeclareVariable(float,    start_attenuation,,"Initial material's attenuation.");
rtDeclareVariable(float,    start_impedance,,"Initial material's impedance.");
rtDeclareVariable(float,    start_mu0,, "Initial material's mu0.");
rtDeclareVariable(float,    start_mu1,, "Initial material's mu1.");
rtDeclareVariable(float,    start_sigma,, "Initial material's sigma.");
rtDeclareVariable(int,      start_index,, "Initial material's index.");
rtDeclareVariable(uint,     launch_id, rtLaunchIndex,);
rtDeclareVariable(uint,     samples_te,, "How many samples per ray are casted.");
rtDeclareVariable(float,    max_travel_time,, );
rtDeclareVariable(uint,     transducer_elements,,);
rtDeclareVariable(float3,   spacing,,);
rtBuffer<float3> transducer_positions_buffer;   //Buffer with Transducer Elements positions
rtBuffer<float3> transducer_directions_buffer;  //Buffer with Transducer Elements directions
rtBuffer<int> results;  //Buffer with distances and BTTIs


rtBuffer<float4, 2> output_buffer;
rtBuffer<ray_physics::segment> segments_buffer;



RT_PROGRAM void raygen(){
    float3 te_pos = transducer_positions_buffer[launch_id];
    float3 te_dir = transducer_directions_buffer[launch_id];
    //if (launch_id == 255u)
    //    rtPrintf("Pos: %f,%f,%f - Dir: %f, %f, %f\n", te_pos.x,te_pos.y,te_pos.z,te_dir.x,te_dir.y,te_dir.z);
    unsigned int sample = 0;
    unsigned int index;
    float r_length;
    float3 to;
    do{
        PRD prd;
        prd.origin = te_pos;
        prd.direction = te_dir;
        prd.importance = 1.f; //attenuation
        prd.depth = 0;
        prd.sample = sample;
        prd.distance_traveled = 0; //[mm]
        prd.time_traveled = 0;
        prd.media_attenuation = start_attenuation; //
        prd.media_impedance = start_impedance; //
        prd.media_mu0 = start_mu0;
        prd.media_mu1 = start_mu1;
        prd.media_sigma = start_sigma;
        prd.btti = 0.f;
        prd.mesh_index = EMPTY_STACK;

        float4 attenuation_impedance_stack[10]; //.x -> attenuation | .y -> impedance | .z -> is_vascular | .w -> material_index
        float4 mu_sigma_stack[10];              //.x -> mu0 | .y -> mu1 | .z -> sigma | .w -> material_index
        int stack_index = 0;
        attenuation_impedance_stack[stack_index] = make_float4(start_attenuation,start_impedance,0.f,EMPTY_STACK);


        PRD old_prd = prd;
        for(;;)         //Iteratively trace rays. When the loop ends, its next bounce will be traced.
        {
            //Check constraints.
            if ((prd.depth >= max_depth) || (prd.importance < scene_epsilon ))
                break;
            //Save old prd to posterior ray marching (Check if assign is Copy or Reference)
            old_prd = prd;
            //Create ray and trace it
            Ray ray = make_Ray(prd.origin, prd.direction, 0, scene_epsilon, RT_DEFAULT_MAX);
            r_length = 10.f /*<- cm to mm*/ * log(scene_epsilon/prd.importance) / attenuation_impedance_stack[stack_index].x * transducer_frequency;
            to = prd.origin + r_length / 100.f * make_float3(spacing.x * prd.direction.x,
                                                             spacing.y * prd.direction.y,
                                                             spacing.z * prd.direction.z); //Para guardar el segmento si no choca con nada mas


            rtTrace(top_object, ray, prd);
            results[launch_id] = results[launch_id]+1;



            index = launch_id + transducer_elements * (prd.depth + max_depth * sample);
            //rtPrintf("Index: %d\n", index);


            if ((old_prd.origin.x == prd.origin.x) && (old_prd.origin.y == prd.origin.y) && (old_prd.origin.z == prd.origin.z)){  //chequea si no se actualizó el origen, quiere decir que no se chocó con nada. Quizas podria estar en un programa MISS
                segments_buffer[index] = ray_physics::segment{       //creo el segmento
                        old_prd.origin,
                        to,//old_prd.origin + max_travel_time * old_prd.direction,
                        old_prd.direction,
                        0.0,//prd.btti,
                        0.0,//old_prd.importance,
                        attenuation_impedance_stack[stack_index].x,
                        mu_sigma_stack[stack_index].x,
                        mu_sigma_stack[stack_index].y,
                        mu_sigma_stack[stack_index].z,
                        old_prd.distance_traveled,
                        launch_id,
                        prd.depth,
                        prd.sample,
                        false
            };
            } else {
                //if (prd.is_vascular == 0.f){
                segments_buffer[index] = ray_physics::segment{       //creo el segmento
                        old_prd.origin,
                        prd.origin,
                        old_prd.direction,
                        prd.btti,
                        old_prd.importance,
                        attenuation_impedance_stack[stack_index].x,
                        mu_sigma_stack[stack_index].x,
                        mu_sigma_stack[stack_index].y,
                        mu_sigma_stack[stack_index].z,
                        old_prd.distance_traveled,
                        launch_id,
                        prd.depth,
                        prd.sample,
                        false
                };/*} else {
                    segments_buffer[index] = ray_physics::segment{       //creo el segmento
                            old_prd.origin,
                            prd.origin,
                            old_prd.direction,
                            0.1f, //para definir manualmente el brillo de las paredes de los vasos
                            old_prd.importance,
                            attenuation_impedance_stack[stack_index].x,
                            mu_sigma_stack[stack_index].x,
                            mu_sigma_stack[stack_index].y,
                            mu_sigma_stack[stack_index].z,
                            old_prd.distance_traveled,
                            launch_id,
                            prd.depth,
                            prd.sample,
                            false
                    };
                }*/
            }



            //Update depth value

            if ((prd.bounce_or_transmission) && (stack_index >= 0)){
                if ((prd.mesh_index == attenuation_impedance_stack[stack_index].w) && (prd.mesh_index != EMPTY_STACK)){ //choco el mismo mesh, entonces estoy saliendo
                    attenuation_impedance_stack[stack_index] = make_float4(0.f);
                    mu_sigma_stack[stack_index] = make_float4(0.f);
                    stack_index = stack_index-1;  //desapilo del stack.
                    //rtPrintf("", stack_index);

                    //if (launch_id == 128) rtPrintf("Saliendo del mesh - %d.\n", stack_index);
                    //if (launch_id == 128)
                    //rtPrintf("Stack index after sub: %d\n", stack_index);
                    //if (launch_id == 256) rtPrintf("Launch id: %d - Index: %d\n", launch_id, stack_index);
                } else {

                    if ((prd.mesh_index == attenuation_impedance_stack[stack_index-1].w) && (prd.mesh_index != EMPTY_STACK)) { //choco con el mesh que estaba afuera

                        attenuation_impedance_stack[stack_index-1] = attenuation_impedance_stack[stack_index]; //saco el material externo
                        mu_sigma_stack[stack_index-1] = mu_sigma_stack[stack_index];                           //y pongo el material que estaba
                        stack_index = stack_index-1;
                        //if (stack_index <= 0) rtPrintf("Estoy dentro de un vaso y estoy saliendo del material que lo rodea - %d.\n", stack_index);

                        //if (launch_id == 500) rtPrintf("Launch id: %d - Index: %d\n", launch_id, stack_index);

                        //if (launch_id == 128)
                        //rtPrintf("Stack index after sub: %d\n", stack_index);
                        //en el tope
                    } else { //choco con un nuevo mesh, estoy entrando.
                        if (attenuation_impedance_stack[stack_index].z){ //si estaba en un vaso
                            stack_index = stack_index+1;

                            //copio el vaso al tope
                            attenuation_impedance_stack[stack_index] = attenuation_impedance_stack[stack_index-1];
                            mu_sigma_stack[stack_index] = mu_sigma_stack[stack_index-1];

                            //agregar el nuevo abajo
                            attenuation_impedance_stack[stack_index-1].x = prd.media_attenuation;//prd.media_impedance,0.f,prd.mesh_index);
                            attenuation_impedance_stack[stack_index-1].y = prd.media_impedance;
                            attenuation_impedance_stack[stack_index-1].z = prd.is_vascular; //AGREGAR IS VASCULAR A LA PRD
                            attenuation_impedance_stack[stack_index-1].w = static_cast<float>(prd.mesh_index);
                            mu_sigma_stack[stack_index-1].x = prd.media_mu0;
                            mu_sigma_stack[stack_index-1].y = prd.media_mu1;
                            mu_sigma_stack[stack_index-1].z = prd.media_sigma;

                        } else {    //si no estaba en un vaso
                            stack_index = stack_index+1;  //apilo en el stack.
                            attenuation_impedance_stack[stack_index].x = prd.media_attenuation;//prd.media_impedance,0.f,prd.mesh_index);
                            attenuation_impedance_stack[stack_index].y = prd.media_impedance;
                            attenuation_impedance_stack[stack_index].w = static_cast<float>(prd.mesh_index);
                            mu_sigma_stack[stack_index].x = prd.media_mu0;// prd.media_mu1, prd.media_sigma, prd.mesh_index);
                            mu_sigma_stack[stack_index].y = prd.media_mu1;
                            mu_sigma_stack[stack_index].z = prd.media_sigma;
                        }
                    }
                }
            }

/*if (launch_id == 100){
                for (int i = 0; i <= 10; i++){
                    rtPrintf("%f - ", attenuation_impedance_stack[i].w);
                }
                rtPrintf("\n");}*/


            //if (launch_id == 128){
            //rtPrintf("stack_index: %d\n", stack_index);
            //rtPrintf("mesh_id: %d\n", attenuation_impedance_stack[stack_index].w);}
            //if (launch_id == 255) rtPrintf("stack handled\n");
            prd.depth++;
        }
        sample++;
        stack_index = 0;
        //if (launch_id == 256) rtPrintf("Launch id: %d - Index: %d\n", launch_id, stack_index);
        //if (launch_id == 512) rtPrintf("------ new frame ------");
    } while( sample < samples_te);

    /*//uint2 out_index;
    //uint3 dist_index;
    //uint3 imp_index;
    //rtPrintf("Distance: %f\n",prd.distance_traveled);
    //rtPrintf("Importance: %f\n", prd.importance);
    /*for (int i = 0; i < max_depth; i++){

        for (int sample = 0; sample < samples_te; sample++){
            dist_index = make_uint3(launch_id*max_depth+i, 0, sample);
            imp_index  = make_uint3(launch_id*max_depth+i, 1, sample);
            //if (results[imp_index] < intensity_epsilon)
            //    break;
            //if (launch_id == 255u)
            //    rtPrintf("Time elapsed: %f\n", results[dist_index]);
            float micros_traveled = results[dist_index];
            float row = micros_traveled / (axial_resolution / speed_of_sound);
            out_index  = make_uint2(launch_id, static_cast<unsigned int>(row)); //static_cast<unsigned int>(prd.distance_traveled));
            //rtPrintf("Micros_traveled: %f - Row: %f\n", micros_traveled, row);
            //rtPrintf("Index: %d, %d\n", out_index.x, out_index.y);
            //rtPrintf("Out_index: %d,%d - Imp_index: %d,%d - Importance: %f\n", out_index.x, out_index.y, imp_index.x, imp_index.y, results[imp_index]);
            if (out_index.y < max_rows){
                output_buffer[out_index] += make_float4(results[imp_index]);
            } else {
                output_buffer[make_uint2(launch_id, max_rows)] += make_float4(results[imp_index]);
            }

            results[imp_index] = 0;
        }

    }*/


}


RT_PROGRAM void exception(){
    const unsigned int code = rtGetExceptionCode();
    /*
    if ( code == RT_EXCEPTION_STACK_OVERFLOW )
        rtPrintf("La cague aca\n");

        //output_buffer[launch_id,static_cast<unsigned int>(prd.distance_traveled)] = error;
    else*/
    rtPrintExceptionDetails();
    //rtPrintExceptionDetails();


}


