#include "hip/hip_runtime.h"
#include "rfimage.h"

using rf_image_ = rf_image<512, 100, 322>;

__global__ __device__
void rf_image_::cuda_convolve(float* axial_kernel, float* lateral_kernel){

}

void rf_image_::cuda_convolve_wrapper(float* axial_kernel, float* lateral_kernel, int axial_size, int lateral_size)
{
    float* dev_axial_kernel;
    float* dev_lateral_kernel;
    //cv::Mat dev_intensities;
    hipMalloc( (void**)&dev_axial_kernel, sizeof(float)*axial_size );
    hipMalloc( (void**)&dev_lateral_kernel, sizeof(float)*lateral_size);

    hipMemcpy( dev_axial_kernel, axial_kernel, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy( dev_lateral_kernel, lateral_kernel, sizeof(float), hipMemcpyHostToDevice);
    //COPIAR DEV_INTENSITIES

    cuda_convolve<<<1,1>>>(axial_kernel, lateral_kernel);

    hipFree(dev_axial_kernel);
    hipFree(dev_lateral_kernel);
}
