
#include <hip/hip_runtime.h>
//#ifndef RANDOM_H
//#define RANDOM_H

//#endif // RANDOM_H

//#include <stdio.h>
//#include <stdlib.h>
//#include <cuda.h>
//#include <curand_kernel.h>
//#include "device_launch_parameters.h"

//#define CUDA_CALL(x) do { if((x) != cudaSuccess) { \
//    printf("Error at %s:%d\n",__FILE__,__LINE__); \
//    }} while(0)

//__global__ void setup_kernel(curandState *state)
//{
//    int id = threadIdx.x + blockIdx.x * 64;
//    /* Each thread gets same seed, a different sequence number,
//       no offset */
//    curand_init(1234, id, 0, &state[id]);
//}
//__global__ float generate_kernel(curandState *state, int *result)
//{
//    int id = threadIdx.x + blockIdx.x * 64;
//    int count = 0;
//    float x;
//    /* Copy state to local memory for efficiency */
//            curandState localState = state[id];
//    /* Generate pseudo-random unsigned ints */
//    for(int n = 0; n < 100000; n++) {
//        x = curand_uniform(&localState);
//        /* Check if low bit set */

//    }
//    /* Copy state back to global memory */
//    state[id] = localState;
//    /* Store results */
//    result[id] += count;
//}



//__host__ void generate_random_numbers(float randoms[], unsigned int size)
//{
//    curandState *devStates;
//    int *devResults, *hostResults;


//    /* Allocate space for results on host */

//    //hostResults = static_cast<int*>(calloc(size, sizeof(int)));

//    /* Allocate space for results on device */

//    CUDA_CALL(cudaMalloc((void **)&devResults, size * sizeof(float)));

//    /* Set results to 0 */

//    CUDA_CALL(cudaMemset(devResults, 0, size * sizeof(float)));

//    /* Allocate space for prng states on device */

//    CUDA_CALL(cudaMalloc((void **)&devStates, size *
//                         sizeof(curandState)));

//    /* Setup prng states */
//    setup_kernel<<<1024, 1024>>>(devStates);
//    /* Generate and use pseudo-random */
//    generate_kernel<<<1024, 1024>>>(devStates, devResults);

//    /* Copy device memory to host */
//    CUDA_CALL(cudaMemcpy(randoms, devResults, size *
//                         sizeof(float), cudaMemcpyDeviceToHost));
//    /* Show result */



//    /* Cleanup device*/
//    CUDA_CALL(cudaFree(devStates));
//    CUDA_CALL(cudaFree(devResults));
//}
