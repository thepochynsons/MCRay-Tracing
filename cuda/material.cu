#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

/*rtDeclareVariable(float, impedance, , );
rtDeclareVariable(float, attenuation_factor, , );
rtDeclareVariable(float, mu0, , );
rtDeclareVariable(float, mu1, , );
rtDeclareVariable(float, sigma, , );
rtDeclareVariable(float, specularity, , );
rtDeclareVariable(float, shininess, , );
rtDeclareVariable(float, thickness, , );*/

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

RT_PROGRAM void closest_hit()
{
    float3 world_shading_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal) );
    float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal) );

    float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );
    compute_ray( );


}
